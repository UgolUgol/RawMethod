#include "hip/hip_runtime.h"
#include "glutWrapper.h"
#include <iostream>

 //======================KERNESL=================================

//================================================================
GlutWrapper* GlutWrapper::class_ptr = nullptr;

GlutWrapper::GlutWrapper(){

}

GlutWrapper::GlutWrapper(int argc, char** argv, int w, int h,
 const char* name){
	this->w = w;
	this->h = h;
	this->name = name;
	this->dt = 0.01;
	class_ptr = this;
	xc = 0.0, yc = 0.0, sx = 10.0, sy = sx * h / w;

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(this->w, this->h);
	glutCreateWindow(this->name);
}

void GlutWrapper::display(){
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(w, h, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	glutSwapBuffers();
}


void GlutWrapper::update_callback(){
	class_ptr->update();
}

void GlutWrapper::display_callback(){
	class_ptr->display();
}

void GlutWrapper::glutRunningFuncs(){
	glutIdleFunc(update_callback);	
	glutDisplayFunc(display_callback);
}



void GlutWrapper::glutSetProjection(GLenum mode){
	glMatrixMode(mode);
	glLoadIdentity();
	gluOrtho2D(0.0, (GLdouble)w, 0.0, (GLdouble)h);	
	glewInit();
}


void GlutWrapper::glutRunSession(){
	buffer = new CudaBuffer(w * h * sizeof(uchar4), GL_PIXEL_UNPACK_BUFFER_ARB);
}


void GlutWrapper::renderCycle(){
	glutRunningFuncs();
	glutSetProjection(GL_PROJECTION);
	glutRunSession();
	glutMainLoop();
}

GlutWrapper::~GlutWrapper(){

}
