#include "hip/hip_runtime.h"
#include "glutWrapper.h"
#include <iostream>

 //======================KERNESL=================================

//================================================================
GlutWrapper* GlutWrapper::class_ptr = nullptr;

GlutWrapper::GlutWrapper(){

}

GlutWrapper::GlutWrapper(int argc, char** argv, int w, int h,
 const char* name) : Controller(w, h) {
	this->name = name;
	class_ptr = this;

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(this->w, this->h);
	glutCreateWindow(this->name);
}

void GlutWrapper::display(){
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(w, h, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	glutSwapBuffers();
}

void GlutWrapper::update_callback(){
	class_ptr->update();
}

void GlutWrapper::display_callback(){
	class_ptr->display();
}

void GlutWrapper::keys_callback(unsigned char key, int x, int y){

// run controller
	class_ptr->keys(key, x, y);

}


void GlutWrapper::glutRunningFuncs(){
	glutIdleFunc(update_callback);	
	glutDisplayFunc(display_callback);
	glutKeyboardFunc(keys_callback);
}



void GlutWrapper::glutSetProjection(GLenum mode){
	glMatrixMode(mode);
	glLoadIdentity();
	gluOrtho2D(0.0, (GLdouble)w, 0.0, (GLdouble)h);	
	glewInit();
}


void GlutWrapper::glutRunSession(){
	buffer = new CudaBuffer(w * h * sizeof(uchar4), GL_PIXEL_UNPACK_BUFFER_ARB);
}


void GlutWrapper::renderCycle(){
	glutRunningFuncs();
	glutSetProjection(GL_PROJECTION);
	glutRunSession();
	glutMainLoop();
}


GlutWrapper::~GlutWrapper(){

}
