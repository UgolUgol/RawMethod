#include "hip/hip_runtime.h"
#include "glutWrapper.h"
#include <iostream>

 //======================KERNESL=================================

<<<<<<< HEAD
__device__ double ff(double x, double y, double t) {
	return sin(x * x + t) + cos(y * y + 0.6 * t) + sin(x * x + y * y + 0.3 * t);
}

__device__ double fun(int i, int j, double t, int w, int h) {
	double xc = 0.0, yc = 0.0, sx = 5.0, sy = sx * h / w;
	double x = 2.0 * i / (double)(w - 1) - 1.0;
	double y = 2.0 * j / (double)(h - 1) - 1.0;
	return ff(xc + sx * x, yc + sy * y, t);
}

__global__ void kernel(uchar4 *data, double t, int w, int h) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;
	int i, j;
	double f;
	for(i = idx; i < w; i += offsetx)
		for(j = idy; j < h; j += offsety) {
			f = (fun(i, j, t, w, h) - minf) / (maxf - minf) * 255;
			data[j * w + i] = make_uchar4(0,(int)f, 0, 255);
		}
}


// ================================================================
/*GlutWrapper* class_ptr;
extern "C"{
	void update_callback(){
		class_ptr->update();
	}

	void display_callback(){
		class_ptr->display();
	}
}
*/

// =============================================

VertexBuffer* vbo;
CudaBuffer* buffer = nullptr;
=======
//================================================================
>>>>>>> beta
GlutWrapper* GlutWrapper::class_ptr = nullptr;

// ==============================================

GlutWrapper::GlutWrapper(){

}

GlutWrapper::GlutWrapper(int argc, char** argv, int w, int h,
 const char* name){
	this->w = w;
	this->h = h;
	this->name = name;
	this->dt = 0.01;
	class_ptr = this;
	xc = 0.0, yc = 0.0, sx = 5.0, sy = sx * h / w;

	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGBA);
	glutInitWindowSize(this->w, this->h);
	glutCreateWindow(this->name);
}

<<<<<<< HEAD
void GlutWrapper::createVertexBuffer(){
	vbo = new VertexBuffer();

	vbo->bind(GL_PIXEL_PACK_BUFFER_ARB);
	vbo->setData(w*h, NULL, GL_DYNAMIC_DRAW);
	vbo->unbind();
}


void GlutWrapper::update(){
	static double t = 0.0;
	uchar4 *data;
	size_t size;
	buffer->mapResource();
	buffer->mappedPointer(&data, size);	
	kernel<<<dim3(32,32), dim3(8,32)>>>(data, t, w, h);
	CSC(hipGetLastError());
	buffer->unmapResource();

	t += 0.05;
	glutPostRedisplay();
}



=======
>>>>>>> beta
void GlutWrapper::display(){
	glClearColor(0.0, 0.0, 0.0, 1.0);
	glClear(GL_COLOR_BUFFER_BIT);
	glDrawPixels(w, h, GL_RGBA, GL_UNSIGNED_BYTE, 0);
	glutSwapBuffers();
}


void GlutWrapper::update_callback(){
	class_ptr->update();
}

void GlutWrapper::display_callback(){
	class_ptr->display();
}

void GlutWrapper::glutRunningFuncs(){
	glutIdleFunc(update_callback);	
	glutDisplayFunc(display_callback);
}



void GlutWrapper::glutSetProjection(GLenum mode){
	glMatrixMode(mode);
	glLoadIdentity();
	gluOrtho2D(0.0, (GLdouble)w, 0.0, (GLdouble)h);	
	glewInit();
}


void GlutWrapper::glutRunSession(){
<<<<<<< HEAD
	createVertexBuffer();
	buffer = new CudaBuffer(vbo, GL_ARRAY_BUFFER);
=======
	buffer = new CudaBuffer(w * h * sizeof(uchar4), GL_PIXEL_UNPACK_BUFFER_ARB);
>>>>>>> beta
}


void GlutWrapper::renderCycle(){
	glutRunningFuncs();
	glutSetProjection(GL_PROJECTION);
	glutRunSession();
	glutMainLoop();
}

GlutWrapper::~GlutWrapper(){

}
