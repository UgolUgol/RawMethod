#include "hip/hip_runtime.h"
#include "swarm.h"
#include "kernels.h"
#include <iostream>
using namespace std;



Swarm::Swarm(int n, int argc, char** argv): objects_count(n), GlutWrapper(argc, argv, 1024, 648, "App"){
// change count to closest squad
	correctParticlesCount();
	device_objects.resize(objects_count);
	forces.resize(objects_count);

// create generator for speed/position
// we need two generators, one for start position and r1 parameter
// second for r2 parameter
	generator = new UniformDist(sq_count);
	help_generator = new UniformDist(sq_count);

// create global optimum start value equal to 20(max val of my function in lim)
	global_opt = thrust::device_malloc<vec2D>(1);
	global_opt[0] = vec2D(5, 5);
	vec2D* g = thrust::raw_pointer_cast(global_opt);

// create device object ptr and generator on device
	Particle* data = thrust::raw_pointer_cast(&device_objects[0]);
	UniformDist* dev_gen;

// generate parameters
	hipMalloc(&dev_gen, sizeof(UniformDist));
	hipMemcpy(dev_gen, generator, sizeof(UniformDist), hipMemcpyHostToDevice);
	generate_parameters<<<sq_count, sq_count>>>(data, objects_count, dev_gen, g);

}

vec2D Swarm::findMassCenter(){

// copy device to host
	host_objects = device_objects;

// mass center vector
	vec2D m = vec2D(0, 0);

// sum coordinates of particles
	for(int i = 0; i < objects_count; i++){
		m = m + host_objects[i].position;
	}

	return 1./(objects_count) * m;
}


void Swarm::correctParticlesCount(){

// this function change value to closest quad
// tnen we can use kernel as <quad, quad>
	sq_count = sqrt(objects_count);

	if(sq_count * sq_count < objects_count){
		sq_count++;
	}
	objects_count = sq_count * sq_count;
}

void Swarm::run(){
	renderCycle();
}

void Swarm::fillBackground(uchar4* data) {
// find center of mass
	vec2D mc = findMassCenter();

// set camera position to center of mass
	setCamera(mc);
	cout<<mc;

// fill background
	background<<<dim3(32,32), dim3(8,32)>>>(data, w, h, xc, yc, sx, sy);
}

void Swarm::addPoints(uchar4* data){

// add points on display
	Particle* points = thrust::raw_pointer_cast(&device_objects[0]);
	drawParticles<<<sq_count, sq_count>>>(data, points, w, h, xc, yc, sx, sy);

}

void Swarm::regenPoints(){

// cast pointer on device object
	Particle* points = thrust::raw_pointer_cast(&device_objects[0]);
	UniformDist* dev_gen;
	UniformDist* dev_hgen;

// copy generators to gpu
	hipMalloc(&dev_gen, sizeof(UniformDist));
	hipMalloc(&dev_hgen, sizeof(UniformDist));
	hipMemcpy(dev_gen, generator, sizeof(UniformDist), hipMemcpyHostToDevice);
	hipMemcpy(dev_hgen, help_generator, sizeof(UniformDist), hipMemcpyHostToDevice);

// device ptr with global min convert to raw pointer
	vec2D* g = thrust::raw_pointer_cast(global_opt);

// calculate forces
	vec2D* fptr = thrust::raw_pointer_cast(&forces[0]);
	forceCalculate<<<sq_count, sq_count>>>(points, objects_count, fptr);

// calculate new position
	regenerate<<<sq_count, sq_count>>>(points, objects_count, fptr, dev_gen, dev_hgen, g, dt);

}

void Swarm::update() {

// start inits 
	static double t = 0.0;
	uchar4 *data;
	size_t size;
	buffer->mapResource();
	buffer->mappedPointer(&data, size);

// draw function contour
	fillBackground(data);

// add points to display
	addPoints(data);

// next step
	regenPoints();

// unbind array
	CSC(hipGetLastError());
	buffer->unmapResource();

	t += dt;
	glutPostRedisplay();
}

double Swarm::F(vec2D vec){
	return -20. * exp(-.2 * sqrt(.5 * (pow(vec.x, 2) + pow(vec.y, 2) ) ) ) - 
			exp(.5 * (cos(2 * pi * vec.x) + cos(2 * pi * vec.y))) + e + 20;
}


Swarm::~Swarm(){

}

