#include "hip/hip_runtime.h"
#include "swarm.h"
#include "kernels.h"
#include <iostream>
using namespace std;



Swarm::Swarm(int n, int argc, char** argv): objects_count(n), GlutWrapper(argc, argv, 1024, 648, "App") {
// change count to closest squad
	correctParticlesCount();
	device_objects.resize(objects_count);
	forces.resize(objects_count);

// create generator for speed/position
	generator = new UniformDist(sq_count);

// create global optimum start value equal to 20(max val of my function in lim)
	global_opt = thrust::device_malloc<vec2D>(1);
	global_opt[0] = vec2D(5, 5);
	vec2D* g = thrust::raw_pointer_cast(global_opt);

// create device object ptr and generator on device
	Particle* data = thrust::raw_pointer_cast(&device_objects[0]);
	UniformDist* dev_gen;

// generate parameters
	hipMalloc(&dev_gen, sizeof(UniformDist));
	hipMemcpy(dev_gen, generator, sizeof(UniformDist), hipMemcpyHostToDevice);
	generate_parameters<<<sq_count, sq_count>>>(data, objects_count, dev_gen, g);

}

void Swarm::correctParticlesCount(){
	sq_count = sqrt(objects_count);

	if(sq_count * sq_count < objects_count){
		sq_count++;
	}
	objects_count = sq_count * sq_count;
}

void Swarm::run(){
	renderCycle();
}

void Swarm::fillBackground(uchar4* data) {
	background<<<dim3(32,32), dim3(8,32)>>>(data, w, h, xc, yc, sx, sy);
}

void Swarm::addPoints(uchar4* data){
	Particle* points = thrust::raw_pointer_cast(&device_objects[0]);
	drawParticles<<<sq_count, sq_count>>>(data, points, w, h, xc, yc, sx, sy);
}

void Swarm::regenPoints(){
	Particle* points = thrust::raw_pointer_cast(&device_objects[0]);
	UniformDist* dev_gen;

// copy generator to gpu
	hipMalloc(&dev_gen, sizeof(UniformDist));
	hipMemcpy(dev_gen, generator, sizeof(UniformDist), hipMemcpyHostToDevice);

// device ptr with global min convert to raw pointer
	vec2D* g = thrust::raw_pointer_cast(global_opt);

// calculate forces
	vec2D* fptr = thrust::raw_pointer_cast(&forces[0]);
	forceCalculate<<<sq_count, sq_count>>>(points, objects_count, fptr);

// calculate new position
	regenerate<<<sq_count, sq_count>>>(points, objects_count, fptr, dev_gen, g, dt);

}

void Swarm::update() {
	static double t = 0.0;
	uchar4 *data;
	size_t size;
	buffer->mapResource();
	buffer->mappedPointer(&data, size);

// draw function contour
	fillBackground(data);

// add points to display
	addPoints(data);

// next step
	regenPoints();


	CSC(hipGetLastError());
	buffer->unmapResource();

	t += dt;
	glutPostRedisplay();
}

double Swarm::F(vec2D vec){
	return -20. * exp(-.2 * sqrt(.5 * (pow(vec.x, 2) + pow(vec.y, 2) ) ) ) - 
			exp(.5 * (cos(2 * pi * vec.x) + cos(2 * pi * vec.y))) + e + 20;
}


Swarm::~Swarm(){

}

