#include "hip/hip_runtime.h"
#include "swarm.h"
#include <iostream>
using namespace std;

Swarm::Swarm(int n): objects_count(n){
	objects.resize(objects_count);
	generator = new UniformDist(-5., 5.);

	for(auto obj = objects.begin(); obj != objects.end(); obj++){
		*obj = Particle();
		vec2D p = (*obj).getLocalOptimum();
		updateOptimum(p);
	}
}


void Swarm::updateOptimum(vec2D p){
	if(F(p) < F(global_opt)){
		global_opt = p;
	}
}


double Swarm::F(vec2D vec){
	return -20. * exp(-.2 * sqrt(.5 * (pow(vec.x, 2) + pow(vec.y, 2) ) ) ) - 
			exp(.5 * (cos(2 * pi * vec.x) + sin(2 * pi * vec.y))) + e + 20;
}


Swarm::~Swarm(){

}

