#include "hip/hip_runtime.h"
#include "swarm.h"
#include "kernels.h"
#include <iostream>
using namespace std;



Swarm::Swarm(int n, int argc, char** argv): objects_count(n), GlutWrapper(argc, argv, 1024, 648, "App") {
	// change count to closest squad
	correctParticlesCount();

	device_objects.resize(objects_count);
	generator = new UniformDist(sq_count);

	Particle* data = thrust::raw_pointer_cast(&device_objects[0]);
	UniformDist* dev_gen;

	hipMalloc(&dev_gen, sizeof(UniformDist));
	hipMemcpy(dev_gen, generator, sizeof(UniformDist), hipMemcpyHostToDevice);
	generate_parameters<<<sq_count, sq_count>>>(data, objects_count, dev_gen);

	host_objects = device_objects;
}

void Swarm::correctParticlesCount(){
	sq_count = sqrt(objects_count);

	if(sq_count * sq_count < objects_count){
		sq_count++;
	}
	objects_count = sq_count * sq_count;
}

void Swarm::run(){
	renderCycle();
}

void Swarm::update() {
	static double t = 0.0;
	uchar4 *data;
	size_t size;
	buffer->mapResource();
	buffer->mappedPointer(&data, size);


// draw function contour
	background<<<dim3(32,32), dim3(8,32)>>>(data, w, h, xc, yc, sx, sy);

// add points to display
	Particle* points = thrust::raw_pointer_cast(&device_objects[0]);
	drawParticles<<<sq_count, sq_count>>>(data, points, w, h, xc, yc, sx, sy);

// next step 


	CSC(hipGetLastError());
	buffer->unmapResource();

	t += 0.05;
	glutPostRedisplay();
}

void Swarm::updateOptimum(vec2D p){
	if(F(p) < F(global_opt)){
		global_opt = p;
	}
}


double Swarm::F(vec2D vec){
	return -20. * exp(-.2 * sqrt(.5 * (pow(vec.x, 2) + pow(vec.y, 2) ) ) ) - 
			exp(.5 * (cos(2 * pi * vec.x) + cos(2 * pi * vec.y))) + e + 20;
}


Swarm::~Swarm(){

}

