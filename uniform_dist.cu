#include "hip/hip_runtime.h"
#include "uniform_dist.h"

// ========= KERNELS ============== //

__global__ void initRandomizer(unsigned int seed, hiprandState* state){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}


// ================================ //

UniformDist::UniformDist(size_t count){
	hipMalloc(&state, sizeof(hiprandState) * count * count);
	initRandomizer<<<count, count>>>(time(NULL), state);
}

__device__ vec2D UniformDist::generate(const int max, const int min, int idx){
	double x = hiprand_uniform(&state[idx]);
	double y = hiprand_uniform(&state[idx]);
	const int diff = max - min + fix;
	x = x * diff + min;
	y = y * diff + min;

	return vec2D(x, y);
}