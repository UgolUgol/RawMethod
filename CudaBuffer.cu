#include "hip/hip_runtime.h"
#include "CudaBuffer.h"
#include <iostream>

<<<<<<< HEAD
CudaBuffer::CudaBuffer(VertexBuffer* buf, GLenum targ,
 	unsigned int flags){
	buffer = buf;
	target = targ;

	buffer->bind(target);;
=======
CudaBuffer::CudaBuffer(unsigned size, 
	GLenum targ, unsigned int flags){
	
	buffer = new VertexBuffer();
	target = targ;

	buffer->bind(target);
	buffer->setData(size, NULL, GL_DYNAMIC_DRAW);
>>>>>>> beta
	hipGraphicsGLRegisterBuffer(&resource, buffer->getId(), flags);
	//buffer->unbind();
}

CudaBuffer::~CudaBuffer(){
	hipGraphicsUnregisterResource(resource);
}

bool CudaBuffer::mapResource(hipStream_t stream){
	return hipGraphicsMapResources(1, &resource, stream) == hipSuccess;
}

bool CudaBuffer::unmapResource(hipStream_t stream){
	return hipGraphicsUnmapResources(1, &resource, stream) == hipSuccess;
}


bool CudaBuffer::mappedPointer(uchar4** ptr, size_t& numBytes){
	bool success = (hipGraphicsResourceGetMappedPointer ( (void**)ptr, &numBytes, resource ) == hipSuccess); 
	return success;   
}


GLuint CudaBuffer::getId(){
	return buffer->getId();
}

GLenum CudaBuffer::getTarget(){
	return target;
}

hipGraphicsResource* CudaBuffer::getResource(){
	return resource;
}

