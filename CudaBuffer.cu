#include "hip/hip_runtime.h"
#include "CudaBuffer.h"
#include <iostream>

CudaBuffer::CudaBuffer(VertexBuffer* buf, GLenum targ,
 	unsigned int flags){
	buffer = buf;
	target = targ;

	buffer->bind(target);;
	hipGraphicsGLRegisterBuffer(&resource, buffer->getId(), flags);
	buffer->unbind();
}

CudaBuffer::~CudaBuffer(){
	hipGraphicsUnregisterResource(resource);
}

bool CudaBuffer::mapResource(hipStream_t stream){
	return hipGraphicsMapResources(1, &resource, stream) == hipSuccess;
}

bool CudaBuffer::unmapResource(hipStream_t stream){
	return hipGraphicsUnmapResources(1, &resource, stream) == hipSuccess;
}


bool CudaBuffer::mappedPointer(uchar4** ptr, size_t& numBytes){
	bool success = (hipGraphicsResourceGetMappedPointer ( (void**)ptr, &numBytes, resource ) == hipSuccess); 
	return success;   
}


GLuint CudaBuffer::getId(){
	return buffer->getId();
}

GLenum CudaBuffer::getTarget(){
	return target;
}

hipGraphicsResource* CudaBuffer::getResource(){
	return resource;
}

